#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

// __global__ void kernel(const uint8_t* src, const int loopCnt, uint8_t* dest_1, uint8_t* dest_2, uint8_t* filter_1, uint8_t* filter_2, float* max_1, float* max_2)
// __global__ void kernel(const uint8_t* src, const int loopCnt, uint8_t* dest_1, uint8_t* dest_2, float* max_1, float* max_2)
__global__ void kernel(const uint8_t* src_1,const uint8_t* src_2, const int loopCnt, float* max_1, float* max_2, float* filter_1, float* filter_2, float hf_st1,float hf_cf1,float hf_st2,float hf_cf2)
{
	double HF_ST1 = hf_st1;
	double HF_CF1 = hf_cf1;
	double HF_ST2 = hf_st2;
	double HF_CF2 = hf_cf2;
	double IDT_1 = HF_ST2;
	double IDT_2 = HF_ST2;
	double OMEGA_C_1 = 2 * M_PI * HF_CF2;
	double OMEGA_C_2 = 2 * M_PI * HF_CF2;
	double AMPLFAC_1 = 1 / ((IDT_2 * OMEGA_C_1 / 2) + 1);
	double AMPLFAC_2 = 1 / ((IDT_2 * OMEGA_C_2 / 2) + 1);
	double Y1C_1 = (IDT_2 * OMEGA_C_1 / 2) - 1;
	double Y1C_2 = (IDT_2 * OMEGA_C_2 / 2) - 1;
	// double DT = IDT;

	// printf("HF_ST2 %f",HF_ST2 );
	// printf("HF_CF2 %f",HF_CF2 );
	const UINT taskIdx = threadIdx.x;
	double output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;
	double _max_1 = 0, _max_2 = 0;
	// if (taskIdx == 0)
	// {
	// 	index = 0;
	// } else {

	// }
	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;
		


		// dest_1[realIdx] = src[realIdx * 2];
		// dest_2[realIdx] = src[realIdx * 2 + 1];

		if((realIdx % loopCnt) == 0)
		{
			if(realIdx > 0)
			{
				for (UINT beforeIndex = realIdx - 500; beforeIndex < realIdx; beforeIndex++)
				{
					output_1 = AMPLFAC_1 * (src_1[beforeIndex] - x1_1 - output_1 * Y1C_1);
					output_2 = AMPLFAC_2 * (src_2[beforeIndex] - x1_2 - output_2 * Y1C_2);
					x1_1 = src_1[beforeIndex];
					x1_2 = src_2[beforeIndex];
				}
				// x1_1 = src_1[realIdx-1];
				// x1_2 = src_2[realIdx-1];
				// output_1 = AMPLFAC_1 * (src_1[realIdx-1] - src_1[realIdx-2] - output_1 * Y1C_1);
				// output_2 = AMPLFAC_2 * (src_2[realIdx-1] - src_2[realIdx-2] - output_2 * Y1C_2);
				// printf("x1_1 \n" , x1_1);
				// printf("loopcnt : %d, %d \n", taskIdx,realIdx);
			} else {
				// 여기에 2500만개 before data 100개를 넣으면 됨
				x1_1 = 0;
				x1_2 = 0;
				output_1= 0;
				output_2 = 0;
				// printf("한번만 탈텐데 :%d, %d \n", taskIdx, realIdx);
			}
			// printf("loopcnt : %d, %d", taskIdx,realIdx);
		}
			
		// if((realIdx % loopCnt) == 0)
		// {
			// x1_1 = src[realIdx*2-2];
			// x1_2 = src[realIdx*2-1];
		// }
		output_1 = AMPLFAC_1 * (src_1[realIdx] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * (src_2[realIdx] - x1_2 - output_2 * Y1C_2);
		
		x1_1 = src_1[realIdx];
		x1_2 = src_2[realIdx];

		// filter_1[realIdx] = floor(output_1*1000) /1000;
		// filter_2[realIdx] = floor(output_2*1000) /1000;
		filter_1[realIdx] = output_1;
		filter_2[realIdx] = output_2;
		
		if(filter_1[realIdx] > _max_1) _max_1 = filter_1[realIdx];
		if(filter_2[realIdx] > _max_2) _max_2 = filter_2[realIdx];

		// if(output_1 > _max_1) _max_1 = output_1;
		// if(output_2 > _max_2) _max_2 = output_2;
	}
	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
	//printf("[%f]", max_1[taskIdx]);
	// printf("[%d] \n", taskIdx);
}

// EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, uint8_t* dest_1, uint8_t* dest_2, uint8_t* filter_1, uint8_t* filter_2, float* max_1, float* max_2)
// EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, uint8_t* dest_1, uint8_t* dest_2, float* max_1, float* max_2)
EXPORT int cudaHighPassFilter(const uint8_t* src_1,const uint8_t* src_2, const int cnt, float* max_1, float* max_2, float* filter_1, float* filter_2, float hf_st1,float hf_cf1,float hf_st2, float hf_cf2)
// EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, float* max_1, float* max_2, )
{
	// float _hf_st1 =0;
	// float _hf_cf1 =0;
	// float _hf_st2 =0;
	// float _hf_cf2 =0;
	// _hf_st1 = hf_st1;
	// _hf_cf1 = hf_cf1;
	// _hf_st2 = hf_st2;
	// _hf_cf2 = hf_cf2;
	// HF_ST1 = _hf_st1;
	// HF_CF1 = _hf_cf1;
	// HF_ST2 = _hf_st2;
	// HF_CF2 = _hf_cf2;
	// printf("in cudaHighPassFilter\n");
	uint8_t *dev_src_1 = 0;
	uint8_t *dev_src_2 = 0;
	// uint8_t *dev_dest_1 = 0, *dev_dest_2 = 0;
	float*dev_filter_1 = 0, *dev_filter_2 = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;

	hipError_t status;

	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("start cuda\n");
	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;
	// printf("success hipSetDevice\n");

	status = hipMalloc((void**)&dev_src_1, cnt* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_src_2, cnt * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	// status = hipMalloc((void**)&dev_dest_1, cnt * sizeof(uint8_t));
	// if (isCudaError(status)) goto Exit;
	// status = hipMalloc((void**)&dev_dest_2, cnt * sizeof(uint8_t));
	// if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_filter_1, cnt * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_filter_2, cnt * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	// printf("success hipMalloc\n");

	status = hipMemcpy(dev_src_1, src_1, cnt* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_src_2, src_2, cnt * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	// printf("success hipMemcpy\n");

	kernel<<<1, UNIT_COUNT>>> (dev_src_1,dev_src_2, cnt / UNIT_COUNT, dev_max_1, dev_max_2,dev_filter_1, dev_filter_2,hf_st1,hf_cf1,hf_st2,hf_cf2);
	// kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_dest_1, dev_dest_2, dev_max_1, dev_max_2);
	// kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_dest_1, dev_dest_2, dev_filter_1, dev_filter_2, dev_max_1, dev_max_2);
	if (isCudaError(hipGetLastError())) goto Exit;
	// printf("success kernel\n");

	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;
	// printf("success hipDeviceSynchronize\n");

	// status = hipMemcpy(dest_1, dev_dest_1, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	// if (isCudaError(status)) goto Exit;
	// status = hipMemcpy(dest_2, dev_dest_2, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	// if (isCudaError(status)) goto Exit;
	status = hipMemcpy(filter_1, dev_filter_1, cnt * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(filter_2, dev_filter_2, cnt * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	// printf("success hipMemcpy\n");

Exit:
	hipFree(dev_src_1);
	hipFree(dev_src_2);
	// hipFree(dev_dest_1);
	// hipFree(dev_dest_2);
	hipFree(dev_filter_1);
	hipFree(dev_filter_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);

	return status;
}