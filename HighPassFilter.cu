#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
// #include "Timer.h"
#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

__global__ void kernel(const uint8_t* src, const int loopCnt,const uint8_t * before_data_1, const uint8_t * before_data_2, float* max_1, float* max_2,float* min_1, float* min_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2, const int OFFSET_1, const float SCALE_1, const int OFFSET_2, const float SCALE_2)
{
	const UINT taskIdx = threadIdx.x;
	float output_1 = 0, output_2 = 0;
	float x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	float _min_1 = (float)OFFSET_1;
	float _min_2 = (float)OFFSET_2;

	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;
		
		// 진폭 알고리즘
		if((realIdx % 4) == 0 && realIdx != 0)
		{
			float _srcMax_1 = 0, _srcMax_2 = 0;
			for(UINT srcIndex = realIdx - 4; srcIndex <realIdx; srcIndex++)
			{
				float __src_1 =  abs((OFFSET_1 - src[srcIndex*2])*SCALE_1);
				float __src_2 =  abs((OFFSET_2 - src[srcIndex*2 + 1])*SCALE_2);
				if(__src_1 > _srcMax_1) _srcMax_1 = __src_1;
				if(__src_2 > _srcMax_2) _srcMax_2 = __src_2;
			}
			if(_srcMax_1 < _min_1) _min_1=_srcMax_1;
			if(_srcMax_2 < _min_2) _min_2=_srcMax_2;
		}
		
		// 2500만개의 index 0 에서 바로 전 2500만개의 뒷부분 가져와서 output_1, x1_1
		if(realIdx == 0)
		{
			// 여기에 2500만개의 before data 100개를 넣으면 됨
			for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
				x1_1 = (float)before_data_1[beforeDataIndex];
				x1_2 = (float)before_data_2[beforeDataIndex];
			}
		}

		// 125056 의 배수들에서 그 앞 데이터 -100번째에서 output_1, x1_1
		if((realIdx % loopCnt) == 0 && realIdx != 0)
		{
			for (UINT beforeIndex = realIdx - BEFORE_DATA_COUNT; beforeIndex < realIdx; beforeIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
				x1_1 = (float)src[beforeIndex*2];
				x1_2 = (float)src[beforeIndex*2+1];
			}
		}

		output_1 = AMPLFAC_1 * ((float)src[realIdx*2] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * ((float)src[realIdx*2+1] - x1_2 - output_2 * Y1C_2);
		
		x1_1 =  (float)src[realIdx*2];
		x1_2 =  (float)src[realIdx*2+1];

		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;
	}

	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
	min_1[taskIdx] = _min_1;
	min_2[taskIdx] = _min_2;
}

EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, const uint8_t * before_data_1, const uint8_t * before_data_2,float* max_1, float* max_2, float* min_1, float* min_2,const double hf_st1,const double hf_cf1,const double hf_st2, const double hf_cf2, const int offset_1, const double scale_1, const int offset_2, const double scale_2)
{
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;
	float *dev_min_1 = 0, *dev_min_2 = 0;
	const int OFFSET_1 = offset_1;
	const int OFFSET_2 = offset_2;
	const float SCALE_1 = scale_1;
	const float SCALE_2 = scale_2;
	const double OMEGA_C_1 = 2 * M_PI * hf_cf2; // 각주파수?
	const double OMEGA_C_2 = 2 * M_PI * hf_cf2;
	const double AMPLFAC_1 = 1 / ((hf_st2 * OMEGA_C_1 / 2) + 1);
	const double AMPLFAC_2 = 1 / ((hf_st2 * OMEGA_C_2 / 2) + 1);
	const double Y1C_1 = (hf_st2 * OMEGA_C_1 / 2) - 1;
	const double Y1C_2 = (hf_st2 * OMEGA_C_2 / 2) - 1;

	hipError_t status;

	if (hf_cf1 < hf_st1 || hf_cf2 < hf_st2)
	{
		printf("Cuda Algorithm Value Error");
		goto Exit;
	}
	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;

	// cuda에 데이터 malloc
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;

	// cuda로 데이터 memcpy
	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;

	// cuda로 작동하는 function
	kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_before_data_1, dev_before_data_2, dev_max_1, dev_max_2,dev_min_1, dev_min_2, AMPLFAC_1, AMPLFAC_2, Y1C_1, Y1C_2, OFFSET_1, SCALE_1, OFFSET_2, SCALE_2);
	if (isCudaError(hipGetLastError())) goto Exit;
	// cuda 동기화
	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;

	// cuda데이터를 Host로 memcpy
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_1, dev_min_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_2, dev_min_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);
	hipFree(dev_min_1);
	hipFree(dev_min_2);

	return status;
}