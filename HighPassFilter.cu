#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Timer.h"
#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

__global__ void kernel(double* times,const uint8_t * before_data_1, const uint8_t * before_data_2,const uint8_t* src, const int loopCnt, float* max_1, float* max_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2)
{

	clock_t start = clock(); 
	const UINT taskIdx = threadIdx.x;
	double output_1 = 0, output_2 = 0;
	double x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	// const UINT beforeIdx = taskIdx * loopCnt;

	// if((beforeIdx % loopCnt) == 0)
	// {
	// 	if(beforeIdx == 0)
	// 	{
	// 		// 여기에 2500만개 before data 100개를 넣으면 됨
	// 		for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
	// 		{
	// 			output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
	// 			output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
	// 			x1_1 = (float)before_data_1[beforeDataIndex];
	// 			x1_2 = (float)before_data_2[beforeDataIndex];
	// 		}
	// 	} else {
	// 		for (UINT beforeIndex = beforeIdx - BEFORE_DATA_COUNT; beforeIndex < beforeIdx; beforeIndex++)
	// 		{
	// 			output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
	// 			output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
	// 			x1_1 = (float)src[beforeIndex*2];
	// 			x1_2 = (float)src[beforeIndex*2+1];
	// 		}
	// 	}
	// }

	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;

		if(realIdx == 0)
		{
			// 여기에 2500만개의 before data 100개를 넣으면 됨
			for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
				x1_1 = (float)before_data_1[beforeDataIndex];
				x1_2 = (float)before_data_2[beforeDataIndex];
			}
		} 
		if((realIdx % loopCnt) == 0 && realIdx !=0)
		{
			for (UINT beforeIndex = realIdx - BEFORE_DATA_COUNT; beforeIndex < realIdx; beforeIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
				x1_1 = (float)src[beforeIndex*2];
				x1_2 = (float)src[beforeIndex*2+1];
			}
		}

		output_1 = AMPLFAC_1 * ((float)src[realIdx*2] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * ((float)src[realIdx*2+1] - x1_2 - output_2 * Y1C_2);
		
		x1_1 = (float) src[realIdx*2];
		x1_2 = (float) src[realIdx*2+1];

		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;
	}

	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
	clock_t stop = clock(); 
    times[taskIdx] = (double)((double)(stop - start) / 1000);
}

EXPORT int cudaHighPassFilter(double * times,const uint8_t * before_data_1, const uint8_t * before_data_2,const uint8_t* src, const int cnt, float* max_1, float* max_2, double hf_st1,double hf_cf1,double hf_st2, double hf_cf2)
{
	// clock_t clock();
	// long long int clock64();
	// WGSTest::Timer timer;
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;
	const double OMEGA_C_1 = 2 * M_PI * hf_cf2;
	const double OMEGA_C_2 = 2 * M_PI * hf_cf2;
	const double AMPLFAC_1 = 1 / ((hf_st2 * OMEGA_C_1 / 2) + 1);
	const double AMPLFAC_2 = 1 / ((hf_st2 * OMEGA_C_2 / 2) + 1);
	const double Y1C_1 = (hf_st2 * OMEGA_C_1 / 2) - 1;
	const double Y1C_2 = (hf_st2 * OMEGA_C_2 / 2) - 1;
	double * dev_times=0; 
	// 알고리즘

	hipError_t status;

	if (hf_cf1 < hf_st1 || hf_cf2 < hf_st2)
	{
		printf("Cuda Algorithm Value Error");
		goto Exit;
	}
	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("start cuda\n");
	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;

	// timer.Reset();
    // timer.Start();
	// cuda에 데이터 malloc
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_times, UNIT_COUNT * sizeof(double));
	if (isCudaError(status)) goto Exit;
	// timer.End();
	// timer.Print("hipMalloc");

	// timer.Reset();
    // timer.Start();
	// cuda로 데이터 memcpy
	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	// timer.End();
	// timer.Print("Device로 hipMemcpy");

	// timer.Reset();
    // timer.Start();

	// cuda로 작동하는 function
	// float timess;
	// hipEvent_t start, stop;
	// hipEventCreate(&start);
	// hipEventCreate(&stop);
	// hipEventRecord(start, 0);

	// kernel<<<1, UNIT_COUNT>>> (dev_times,dev_before_data_1, dev_before_data_2,dev_src, cnt / UNIT_COUNT, dev_max_1, dev_max_2,AMPLFAC_1,AMPLFAC_2,Y1C_1,Y1C_2);
	// hipEventRecord(stop);
	// hipEventSynchronize(stop);
	// hipEventElapsedTime(&timess, start, stop);
	// printf("Elapsed time : %f ms\n" ,timess);

	if (isCudaError(hipGetLastError())) goto Exit;

	// cuda 동기화
	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;
	// timer.End();
	// timer.Print("kernel");



	// timer.Reset();
    // timer.Start();
	// cuda데이터를 Host로 memcpy
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(times, dev_times, UNIT_COUNT * sizeof(double), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	// timer.End();
	// timer.Print("Host로 hipMemcpy");

	// for(UINT i = 0; i <200; i++)
	// {
	// 	printf("times %d %d",i, times[i]);
	// }
	// printf("times %d ",times[0]);

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);
	hipFree(dev_times);

	return status;
}