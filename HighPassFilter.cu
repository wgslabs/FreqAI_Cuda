#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
// #include "Timer.h"
#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}
__global__ void kernel(const uint8_t* src, const int loopCnt,const uint8_t * before_data_1, const uint8_t * before_data_2, float* max_1, float* max_2,uint8_t* min_1, uint8_t* min_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2, const int OFFSET_1, const float SCALE_1, const int OFFSET_2, const float SCALE_2, int* ampResults1, int* ampResults2, int ampThreshold, uint8_t *ampMaxs1,uint8_t *ampMaxs2)
// __global__ void kernel(const uint8_t* src, const int loopCnt,const uint8_t * before_data_1, const uint8_t * before_data_2, float* max_1, float* max_2,uint8_t* min_1, uint8_t* min_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2, const int OFFSET_1, const float SCALE_1, const int OFFSET_2, const float SCALE_2)
{
	const UINT taskIdx = threadIdx.x;
	float output_1 = 0, output_2 = 0;
	float x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	uint8_t _min_1 = 255;
	uint8_t _min_2 = 255;
	int ampMaxTotal1 = 0;
	int ampMaxTotal2 = 0;
	int ampMaxsIdx = taskIdx * loopCnt/ AMP_DEVIDE_COUNT; // taskIdx 0~199, loopCnt =125056, 5
	
	for(UINT index = 0; index < loopCnt; index++)
	{
		if(index < AMP_RESULT_DATA_COUNT){
			const int ampResultsIdx = taskIdx * AMP_RESULT_DATA_COUNT + index;
			ampResults1[ampResultsIdx] = -1;
			ampResults2[ampResultsIdx] = -1;
		}
		const UINT realIdx = taskIdx * loopCnt + index;
		
		// 진폭 알고리즘
		if((realIdx % AMP_DEVIDE_COUNT) == 0 && realIdx != 0)
		{
			ampMaxs1[ampMaxsIdx] = 0;
			ampMaxs2[ampMaxsIdx] = 0;
			// uint8_t _srcMax_1 = 0, _srcMax_2 = 0;
			for(UINT srcIndex = realIdx - AMP_DEVIDE_COUNT; srcIndex <realIdx; srcIndex++)
			{
				
				const uint8_t __src_1 =  abs((OFFSET_1 - src[srcIndex*2]));
				const uint8_t __src_2 =  abs((OFFSET_2 - src[srcIndex*2 + 1]));
				if(__src_1 > ampMaxs1[ampMaxsIdx]) ampMaxs1[ampMaxsIdx] = __src_1;
				if(__src_2 > ampMaxs2[ampMaxsIdx]) ampMaxs2[ampMaxsIdx] = __src_2;
				// if(__src_1 > _srcMax_1) _srcMax_1 = __src_1;
				// if(__src_2 > _srcMax_2) _srcMax_2 = __src_2;
			}
			// if(_srcMax_1 < _min_1) _min_1=_srcMax_1;
			// if(_srcMax_1 < _min_1) _min_1=_srcMax_1;
			ampMaxTotal1 = ampMaxTotal1 + ampMaxs1[ampMaxsIdx];
			ampMaxTotal2 = ampMaxTotal2 + ampMaxs2[ampMaxsIdx];
			if(ampMaxs1[ampMaxsIdx] < _min_1) _min_1=ampMaxs1[ampMaxsIdx];
			if(ampMaxs2[ampMaxsIdx] < _min_2) _min_2=ampMaxs2[ampMaxsIdx];
			ampMaxsIdx++;
		}
		
		// 2500만개의 index 0 에서 바로 전 2500만개의 뒷부분 가져와서 output_1, x1_1
		if(realIdx == 0)
		{
			// 여기에 2500만개의 before data 100개를 넣으면 됨
			for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
				x1_1 = (float)before_data_1[beforeDataIndex];
				x1_2 = (float)before_data_2[beforeDataIndex];
			}
		}

		// 125056 의 배수들에서 그 앞 데이터 -100번째에서 output_1, x1_1
		if((realIdx % loopCnt) == 0 && realIdx != 0)
		{
			for (UINT beforeIndex = realIdx - BEFORE_DATA_COUNT; beforeIndex < realIdx; beforeIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
				x1_1 = (float)src[beforeIndex*2];
				x1_2 = (float)src[beforeIndex*2+1];
			}
		}

		output_1 = AMPLFAC_1 * ((float)src[realIdx*2] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * ((float)src[realIdx*2+1] - x1_2 - output_2 * Y1C_2);
		
		x1_1 =  (float)src[realIdx*2];
		x1_2 =  (float)src[realIdx*2+1];

		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;
	}
	
	const float ampMaxAvg1 = ampMaxTotal1/ (loopCnt/AMP_DEVIDE_COUNT);
	const float ampMaxAvg2 = ampMaxTotal2/ (loopCnt/AMP_DEVIDE_COUNT);
	int ampResultIdx1 = 0;
	int ampResultIdx2 = 0;
	for(UINT index = 0; index < loopCnt/AMP_DEVIDE_COUNT; index++){
		const UINT ampMaxsIdx = taskIdx * loopCnt/AMP_DEVIDE_COUNT + index;
		const uint8_t diff1 = 100-((ampMaxs1[ampMaxsIdx] / ampMaxAvg1)  *100);
		const uint8_t diff2 = 100-((ampMaxs2[ampMaxsIdx] / ampMaxAvg2)  *100);
		const UINT ampResultIdx = taskIdx * AMP_RESULT_DATA_COUNT;
		if( ampThreshold < diff1 && ampResultIdx1 < 100){
			ampResults1[ampResultIdx+ampResultIdx1++] = ampMaxsIdx * AMP_DEVIDE_COUNT;
			// printf("1 %d_%d_%d_%f,%d\n",ampResultIdx, ampResultIdx1, ampMaxsIdx * AMP_DEVIDE_COUNT, ampMaxAvg1 , ampMaxs1[ampMaxsIdx]);
		}
		if( ampThreshold < diff2 && ampResultIdx2 < 100){
			ampResults2[ampResultIdx+ampResultIdx2++] = ampMaxsIdx * AMP_DEVIDE_COUNT;
			// printf("2 %d_%d_%d_%f,%d\n",ampResultIdx, ampResultIdx2, ampMaxsIdx * AMP_DEVIDE_COUNT, ampMaxAvg2 , ampMaxs2[ampMaxsIdx]);
		}

	}
	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
	min_1[taskIdx] = _min_1;
	min_2[taskIdx] = _min_2;
}
EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, const uint8_t * before_data_1, const uint8_t * before_data_2,float* max_1, float* max_2, uint8_t* min_1, uint8_t* min_2,const double hf_st1,const double hf_cf1,const double hf_st2, const double hf_cf2, const int offset_1, const double scale_1, const int offset_2, const double scale_2, int* ampResult1, int* ampResult2, const int ampThreshold)
// EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, const uint8_t * before_data_1, const uint8_t * before_data_2,float* max_1, float* max_2, uint8_t* min_1, uint8_t* min_2,const double hf_st1,const double hf_cf1,const double hf_st2, const double hf_cf2, const int offset_1, const double scale_1, const int offset_2, const double scale_2)
{
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;
	uint8_t *dev_min_1 = 0, *dev_min_2 = 0;
	int *dev_ampResults1 = 0, *dev_ampResults2 = 0;
	const int OFFSET_1 = offset_1;
	const int OFFSET_2 = offset_2;
	const float SCALE_1 = scale_1;
	const float SCALE_2 = scale_2;
	const double OMEGA_C_1 = 2 * M_PI * hf_cf2; // 각주파수?
	const double OMEGA_C_2 = 2 * M_PI * hf_cf2;
	const double AMPLFAC_1 = 1 / ((hf_st2 * OMEGA_C_1 / 2) + 1);
	const double AMPLFAC_2 = 1 / ((hf_st2 * OMEGA_C_2 / 2) + 1);
	const double Y1C_1 = (hf_st2 * OMEGA_C_1 / 2) - 1;
	const double Y1C_2 = (hf_st2 * OMEGA_C_2 / 2) - 1;
	
	uint8_t *dev_ampMaxs1 =0;
	uint8_t *dev_ampMaxs2 =0;
	hipError_t status;

	if (hf_cf1 < hf_st1 || hf_cf2 < hf_st2)
	{
		printf("Cuda Algorithm Value Error");
		goto Exit;
	}
	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;
	// printf("1 \n");
	// cuda에 데이터 malloc
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_1, UNIT_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_2, UNIT_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	
	status = hipMalloc((void**)&dev_ampResults1, UNIT_COUNT * AMP_RESULT_DATA_COUNT* sizeof(uint32_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_ampResults2, UNIT_COUNT * AMP_RESULT_DATA_COUNT* sizeof(uint32_t));
	if (isCudaError(status)) goto Exit;

	status = hipMalloc((void**)&dev_ampMaxs1,  cnt / AMP_DEVIDE_COUNT * sizeof(uint8_t)); //2500만 /5
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_ampMaxs2,  cnt / AMP_DEVIDE_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;

	// cuda로 데이터 memcpy
	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;

	// cuda로 작동하는 function
	// kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_before_data_1, dev_before_data_2, dev_max_1, dev_max_2,dev_min_1, dev_min_2, AMPLFAC_1, AMPLFAC_2, Y1C_1, Y1C_2, OFFSET_1, SCALE_1, OFFSET_2, SCALE_2);
	kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_before_data_1, dev_before_data_2, dev_max_1, dev_max_2,dev_min_1, dev_min_2, AMPLFAC_1, AMPLFAC_2, Y1C_1, Y1C_2, OFFSET_1, SCALE_1, OFFSET_2, SCALE_2, dev_ampResults1, dev_ampResults2, ampThreshold, dev_ampMaxs1, dev_ampMaxs2);
	if (isCudaError(hipGetLastError())) goto Exit;
	// cuda 동기화
	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;

	// cuda데이터를 Host로 memcpy
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_1, dev_min_1, UNIT_COUNT * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_2, dev_min_2, UNIT_COUNT * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;

	// status = hipMemcpy(ampResult1, dev_ampResults1, UNIT_COUNT * AMP_RESULT_DATA_COUNT *sizeof(int), hipMemcpyDeviceToHost);
	// if (isCudaError(status)) goto Exit;
	// status = hipMemcpy(ampResult2, dev_ampResults2, UNIT_COUNT * AMP_RESULT_DATA_COUNT *sizeof(int), hipMemcpyDeviceToHost);
	// if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);
	hipFree(dev_min_1);
	hipFree(dev_min_2);
	hipFree(dev_ampResults1);
	hipFree(dev_ampResults2);
	hipFree(dev_ampMaxs1);
	hipFree(dev_ampMaxs2);

	return status;
}



__global__ void kernel2(const uint8_t* src, const int loopCnt,const uint8_t * before_data_1, const uint8_t * before_data_2, float* max_1, float* max_2,uint8_t* min_1, uint8_t* min_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2, const int OFFSET_1, const float SCALE_1, const int OFFSET_2, const float SCALE_2)
{
	const UINT taskIdx = threadIdx.x;
	float output_1 = 0, output_2 = 0;
	float x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	uint8_t _min_1 = 255;
	uint8_t _min_2 = 255;
	uint8_t _ampMax1 = 0;
	uint8_t _ampMax2 = 0;
	// if(taskIdx ==143){
	// 	printf("%u %u \n",taskIdx * loopCnt , taskIdx * loopCnt +loopCnt);
	// }
	int a = 0;
	for(UINT index = 0; index < loopCnt; index++)
	{
		
		const UINT realIdx = taskIdx * loopCnt + index;

		// if(taskIdx ==143){
		// 	if(abs(OFFSET_1 - src[realIdx*2])==98){
		// 		printf("%u %u \n",realIdx, index);
		// 	}
		// }
		// if(realIdx >= 0 && realIdx <= 10)
		// {
		// 	printf("taskidx %u realIdx %u %d %u \n",taskIdx, realIdx, a, abs(OFFSET_1 - src[realIdx*2]));	
		// }

		// if(realIdx >= 17917050 && realIdx <= 17917070)
		// {
		// 	printf("taskidx %u realIdx %u %d %u \n",taskIdx, realIdx, a, abs(OFFSET_1 - src[realIdx*2]));	
		// }

		// 기존
		// if((realIdx % AMP_DEVIDE_COUNT) == 0 && realIdx != 0)
		// {
		// 	uint8_t _ampMax1 = 0;
		// 	uint8_t _ampMax2 = 0;
		// 	for(UINT srcIndex = realIdx - AMP_DEVIDE_COUNT; srcIndex < realIdx; srcIndex++)
		// 	{
		// 		const uint8_t __src_1 =  abs((OFFSET_1 - src[srcIndex*2]));
		// 		const uint8_t __src_2 =  abs((OFFSET_2 - src[srcIndex*2 + 1]));
		// 		if(__src_1 > _ampMax1) _ampMax1 = __src_1;
		// 		if(__src_2 > _ampMax2) _ampMax2 = __src_2;
		// 	}
		// 	if(_ampMax1 < _min_1) _min_1=_ampMax1;
		// 	if(_ampMax2 < _min_2) _min_2=_ampMax2;
			
		// 	// if(taskIdx ==143){
		// 	// 	if(_ampMax1 == 98){
		// 	// 		printf("realIdx %u %d \n",realIdx, a);
		// 	// 	}
		// 	// }
			
		// 	// if(taskIdx ==143){
		// 	// 	if(a == 6809){
		// 	// 		printf("Taskidx %u realIdx %u %d %u \n",taskIdx, realIdx, a, _ampMax1);
		// 	// 	}
		// 	// 	if(a == 6810){
		// 	// 		printf("Taskidx %u realIdx %u %d %u \n",taskIdx,realIdx, a, _ampMax1);
		// 	// 	}
		// 	// 	if(a == 6811){
		// 	// 		printf("Taskidx %u realIdx %u %d %u \n",taskIdx,realIdx, a, _ampMax1);
		// 	// 	}
		// 	// }
		// 	a++;
		// }
		
		// 2500만개의 index 0 에서 바로 전 2500만개의 뒷부분 가져와서 output_1, x1_1
		if(realIdx == 0)
		{
			// 여기에 2500만개의 before data 100개를 넣으면 됨
			for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
				x1_1 = (float)before_data_1[beforeDataIndex];
				x1_2 = (float)before_data_2[beforeDataIndex];
			}
		}

		// 125056 의 배수들에서 그 앞 데이터 -100번째에서 output_1, x1_1
		if((realIdx % loopCnt) == 0 && realIdx != 0)
		{
			for (UINT beforeIndex = realIdx - BEFORE_DATA_COUNT; beforeIndex < realIdx; beforeIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
				x1_1 = (float)src[beforeIndex*2];
				x1_2 = (float)src[beforeIndex*2+1];
			}
		}
		const uint8_t value1 = abs((OFFSET_1 - src[realIdx*2]));
		const uint8_t value2 = abs((OFFSET_2 - src[realIdx*2+1]));
		output_1 = (float) AMPLFAC_1 * (value1 - x1_1 - output_1 * Y1C_1);
		output_2 = (float) AMPLFAC_2 * (value2 - x1_2 - output_2 * Y1C_2);
		
		x1_1 =  (float)src[realIdx*2];
		x1_2 =  (float)src[realIdx*2+1];
		
		// H ALGO
		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;

		// A ALGO
		if(value1 > _ampMax1) _ampMax1 = value1;
		if(value2 > _ampMax2) _ampMax2 = value2;

		// 진폭 알고리즘
		if((realIdx % AMP_DEVIDE_COUNT) == AMP_DEVIDE_COUNT - 1)
		{
			if(_ampMax1 < _min_1) _min_1=_ampMax1;
			if(_ampMax2 < _min_2) _min_2=_ampMax2;
			_ampMax1 = 0;
			_ampMax2 = 0;
			// count 위해서 남김
			a++;
		}
	}
	//printf("%ud %ud \n",taskIdx, _min_1);
	// if(taskIdx ==143){
		
	// 		printf("aaaaaaaaaaa  %d \n",a);
		
	// }		
	printf("aaaaaaaaaaa  %d, %d \n", taskIdx,a);
	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
	min_1[taskIdx] = _min_1;
	min_2[taskIdx] = _min_2;
}
EXPORT int cudaHighPassFilter2(const uint8_t* src, const int cnt, const uint8_t * before_data_1, const uint8_t * before_data_2,float* max_1, float* max_2, uint8_t* min_1, uint8_t* min_2,const double hf_st1,const double hf_cf1,const double hf_st2, const double hf_cf2, const int offset_1, const double scale_1, const int offset_2, const double scale_2)
{
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;
	uint8_t *dev_min_1 = 0, *dev_min_2 = 0;
	const int OFFSET_1 = offset_1;
	const int OFFSET_2 = offset_2;
	const float SCALE_1 = scale_1;
	const float SCALE_2 = scale_2;
	const double OMEGA_C_1 = 2 * M_PI * hf_cf2;
	const double OMEGA_C_2 = 2 * M_PI * hf_cf2;
	const double AMPLFAC_1 = 1 / ((hf_st2 * OMEGA_C_1 / 2) + 1);
	const double AMPLFAC_2 = 1 / ((hf_st2 * OMEGA_C_2 / 2) + 1);
	const double Y1C_1 = (hf_st2 * OMEGA_C_1 / 2) - 1;
	const double Y1C_2 = (hf_st2 * OMEGA_C_2 / 2) - 1;
	
	hipError_t status;

	if (hf_cf1 < hf_st1 || hf_cf2 < hf_st2)
	{
		printf("Cuda Algorithm Value Error");
		goto Exit;
	}
	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;
	// printf("1 \n");
	// cuda에 데이터 malloc
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_1, UNIT_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_min_2, UNIT_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	
	// cuda로 데이터 memcpy
	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;

	// cuda로 작동하는 function
	kernel2<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_before_data_1, dev_before_data_2, dev_max_1, dev_max_2,dev_min_1, dev_min_2, AMPLFAC_1, AMPLFAC_2, Y1C_1, Y1C_2, OFFSET_1, SCALE_1, OFFSET_2, SCALE_2);
	if (isCudaError(hipGetLastError())) goto Exit;
	// cuda 동기화
	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;

	// cuda데이터를 Host로 memcpy
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_1, dev_min_1, UNIT_COUNT * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(min_2, dev_min_2, UNIT_COUNT * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;

	// if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);
	hipFree(dev_min_1);
	hipFree(dev_min_2);

	return status;
}