#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

__global__ void kernel(const uint8_t * before_data_1, const uint8_t * before_data_2,const uint8_t* src, const int loopCnt, float* max_1, float* max_2, double hf_st1,double hf_cf1,double hf_st2,double hf_cf2)
{
	double HF_ST1 = hf_st1;
	double HF_CF1 = hf_cf1;
	double HF_ST2 = hf_st2;
	double HF_CF2 = hf_cf2;
	double IDT_1 = HF_ST2;
	double IDT_2 = HF_ST2;
	double OMEGA_C_1 = 2 * M_PI * HF_CF2;
	double OMEGA_C_2 = 2 * M_PI * HF_CF2;
	double AMPLFAC_1 = 1 / ((IDT_2 * OMEGA_C_1 / 2) + 1);
	double AMPLFAC_2 = 1 / ((IDT_2 * OMEGA_C_2 / 2) + 1);
	double Y1C_1 = (IDT_2 * OMEGA_C_1 / 2) - 1;
	double Y1C_2 = (IDT_2 * OMEGA_C_2 / 2) - 1;
	// double DT = IDT;

	if (HF_CF1 < HF_ST1 || HF_CF2 < HF_ST2)
	{
		printf("Cuda Algorithm Value Error");
	}


	const UINT taskIdx = threadIdx.x;
	double output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	const UINT beforeIdx = taskIdx * loopCnt;

	if((beforeIdx % loopCnt) == 0)
		{
			if(beforeIdx > 0)
			{
				for (UINT beforeIndex = beforeIdx - 100; beforeIndex < beforeIdx; beforeIndex++)
				{
					output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
					output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
					x1_1 = (float)src[beforeIndex*2];
					x1_2 = (float)src[beforeIndex*2+1];
				}
			} else {
				// 여기에 2500만개 before data 100개를 넣으면 됨
				for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
				{
					output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
					output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
					x1_1 = (float)before_data_1[beforeDataIndex];
					x1_2 = (float)before_data_2[beforeDataIndex];
				}
			}
		}

	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;
		
		output_1 = AMPLFAC_1 * ((float)src[realIdx*2] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * ((float)src[realIdx*2+1] - x1_2 - output_2 * Y1C_2);
		
		x1_1 = (float) src[realIdx*2];
		x1_2 = (float) src[realIdx*2+1];

		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;
	}

	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
}

EXPORT int cudaHighPassFilter(const uint8_t * before_data_1, const uint8_t * before_data_2,const uint8_t* src, const int cnt, float* max_1, float* max_2, double hf_st1,double hf_cf1,double hf_st2, double hf_cf2)
{
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;

	hipError_t status;

	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("start cuda\n");
	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;

	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	
	kernel<<<1, UNIT_COUNT>>> (dev_before_data_1, dev_before_data_2,dev_src, cnt / UNIT_COUNT, dev_max_1, dev_max_2,hf_st1,hf_cf1,hf_st2,hf_cf2);
	
	if (isCudaError(hipGetLastError())) goto Exit;

	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;
	
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);

	return status;
}