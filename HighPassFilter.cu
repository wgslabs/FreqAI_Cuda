#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Timer.h"
#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

__global__ void kernel(const uint8_t* src, const int loopCnt,const uint8_t * before_data_1, const uint8_t * before_data_2, float* max_1, float* max_2,const double AMPLFAC_1,const double AMPLFAC_2,const double Y1C_1,const double Y1C_2)
{
	const UINT taskIdx = threadIdx.x;
	float output_1 = 0, output_2 = 0;
	float x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;

	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;

		if(realIdx == 0)
		{
			// 여기에 2500만개의 before data 100개를 넣으면 됨
			for (UINT beforeDataIndex = 0; beforeDataIndex < BEFORE_DATA_COUNT; beforeDataIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)before_data_1[beforeDataIndex] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)before_data_2[beforeDataIndex] - x1_2 - output_2 * Y1C_2);;
				x1_1 = (float)before_data_1[beforeDataIndex];
				x1_2 = (float)before_data_2[beforeDataIndex];
			}
		} 
		if((realIdx % loopCnt) == 0 && realIdx != 0)
		{
			for (UINT beforeIndex = realIdx - BEFORE_DATA_COUNT; beforeIndex < realIdx; beforeIndex++)
			{
				output_1 = AMPLFAC_1 * ((float)src[beforeIndex*2] - x1_1 - output_1 * Y1C_1);
				output_2 = AMPLFAC_2 * ((float)src[beforeIndex*2+1] - x1_2 - output_2 * Y1C_2);
				x1_1 = (float)src[beforeIndex*2];
				x1_2 = (float)src[beforeIndex*2+1];
			}
		}

		output_1 = AMPLFAC_1 * ((float)src[realIdx*2] - x1_1 - output_1 * Y1C_1);
		output_2 = AMPLFAC_2 * ((float)src[realIdx*2+1] - x1_2 - output_2 * Y1C_2);
		
		x1_1 =  (float)src[realIdx*2];
		x1_2 =  (float)src[realIdx*2+1];

		if(output_1 > _max_1) _max_1 = output_1;
		if(output_2 > _max_2) _max_2 = output_2;
	}

	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
}

EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, const uint8_t * before_data_1, const uint8_t * before_data_2,float* max_1, float* max_2, const double hf_st1,const double hf_cf1,const double hf_st2, const double hf_cf2)
{
	WGSTest::Timer timer;
	uint8_t *dev_before_data_1 = 0;
	uint8_t *dev_before_data_2 = 0;
	uint8_t *dev_src = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;
	const double OMEGA_C_1 = 2 * M_PI * hf_cf2;
	const double OMEGA_C_2 = 2 * M_PI * hf_cf2;
	const double AMPLFAC_1 = 1 / ((hf_st2 * OMEGA_C_1 / 2) + 1);
	const double AMPLFAC_2 = 1 / ((hf_st2 * OMEGA_C_2 / 2) + 1);
	const double Y1C_1 = (hf_st2 * OMEGA_C_1 / 2) - 1;
	const double Y1C_2 = (hf_st2 * OMEGA_C_2 / 2) - 1;

	hipError_t status;

	if (hf_cf1 < hf_st1 || hf_cf2 < hf_st2)
	{
		printf("Cuda Algorithm Value Error");
		goto Exit;
	}
	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;

	// cuda에 데이터 malloc
	status = hipMalloc((void**)&dev_src, (cnt*2)* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;

	// cuda로 데이터 memcpy
	status = hipMemcpy(dev_src, src, (cnt*2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_1, before_data_1, BEFORE_DATA_COUNT* sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dev_before_data_2, before_data_2, BEFORE_DATA_COUNT * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;

	timer.Reset();
    timer.Start();
	// cuda로 작동하는 function
	kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_before_data_1, dev_before_data_2, dev_max_1, dev_max_2, AMPLFAC_1, AMPLFAC_2, Y1C_1, Y1C_2);
	if (isCudaError(hipGetLastError())) goto Exit;
	// cuda 동기화
	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;
	timer.End();
	timer.Print("kernel");

	// cuda데이터를 Host로 memcpy
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_src);
	hipFree(dev_before_data_1);
	hipFree(dev_before_data_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);

	return status;
}